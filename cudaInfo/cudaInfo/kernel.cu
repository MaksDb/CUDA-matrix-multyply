
#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>
#define BLOCK_SIZE  16          // submatrix size
#define N           1024        // matrix size is N*N


hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void matMult(float * a, float * b, int n, float * c)
{
	int   bx = blockIdx.x;     // block index
	int   by = blockIdx.y;
	int   tx = threadIdx.x;        // thread index
	int   ty = threadIdx.y;
	float sum = 0.0f;           // computed subelement
	int   ia = n * BLOCK_SIZE * by + n * ty;   // a [i][0]
	int   ib = BLOCK_SIZE * bx + tx;

	// Multiply the two matrices together;
	for (int k = 0; k < n; k++)
		sum += a[ia + k] * b[ib + k*n];

	// Write the block sub-matrix to global memory;
	// each thread writes one element
	int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;

	c[ic + n * ty + tx] = sum;
}

int main()
{
	int numBytes = N * N * sizeof(float);

	// allocate host memory
	float * a = new float[N*N];
	float * b = new float[N*N];
	float * c = new float[N*N];

	for (int i = 0; i < N; i++)
		for (int j = 0; j < N; j++)
		{
			int	k = N*i + j;

			a[k] = k;
			b[k] = -k;
		}

	// allocate device memory
	float * adev = NULL;
	float * bdev = NULL;
	float * cdev = NULL;

	hipMalloc((void**)&adev, numBytes);
	hipMalloc((void**)&bdev, numBytes);
	hipMalloc((void**)&cdev, numBytes);

	// set kernel launch configuration
	dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocks(N / threads.x, N / threads.y);

	// create cuda event handles
	hipEvent_t start, stop;
	float gpuTime = 0.0f;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	// asynchronously issue work to the GPU (all to stream 0)
	hipEventRecord(start, 0);
	hipMemcpy(adev, a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(bdev, b, numBytes, hipMemcpyHostToDevice);

	matMult <<<blocks, threads >>> (adev, bdev, N, cdev);

	hipMemcpy(c, cdev, numBytes, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);

	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpuTime, start, stop);

	// print the cpu and gpu times
	printf("time spent executing by the GPU: %.2f millseconds\n", gpuTime);

	// release resources
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipFree(adev);
	hipFree(bdev);
	hipFree(cdev);

	delete a;
	delete b;
	delete c;
	system("pause");
	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // cudaDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
